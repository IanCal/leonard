#include "hip/hip_runtime.h"
/*
 *   This file is part of Leonard.
 *
 *   Foobar is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   Foobar is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with Leonard.  If not, see <http://www.gnu.org/licenses/>.
 */

// Need to cut down on the includes, these are the max required, not sure
// which are needed
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include  <sys/timeb.h>
#include <allegro.h>

//mmap stuff
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/mman.h>


/* Includes, cuda */
#include "hipblas.h"
#include "kernels.cu"
#include "rbm.cuh"
#include "ParameterController.cuh"


/* 
 * The actual code for the RBM goes in here
 */

RBM::RBM(int numLayers, int *sizeOfLayers, int *sizeOfLabels, ParameterController *parameterController){
	printf("Creating RBM with \n");
	numberOfWeightLayers=numLayers-1;
	numberOfNeuronLayers=numLayers;
	learningRates = new float[numLayers]; 
	momentum = new float[numLayers];
	parameterUpdater = parameterController; 	
	parameterUpdater->initialise(this);

	// Need to define the device variables
	for( int layer=0 ; layer<numberOfNeuronLayers-1 ; layer++ )
	{
		//set 
		//d_input_t0[layer]
		//d_input_pt0[layer];
		//d_input_tn[layer]
		//d_input_ptn[layer]
		//d_output_t0[layer]=d_input_t0[layer+1];s
		//d_weights
		//d_inputBiases
		//d_outputBiases
		//
	}
	//final out/inputs
	
};

void RBM::pushDown(int layer, bool input_t0, bool output_t0, bool useProbabilities){

	//Basic variables
	
	int inputs = layerSizes[layer];
	int outputs = layerSizes[layer+1];
	int inputBatchSize = inputs * batchSize;
	int numberOfBlocks = inputBatchSize/blockSize + (inputBatchSize%blockSize == 0?0:1);

	//device pointers
	
	float* d_input;
	float* d_input_p;
	float* d_output;
	if (input_t0){
		d_input_p = d_input_pt0[layer];
		d_input = d_input_t0[layer];	
	}
	else{
		d_input_p = d_input_ptn[layer];
		d_input = d_input_tn[layer];
	}

	if (output_t0){
		if (useProbabilities)
			d_output = d_output_pt0[layer];
		else
			d_output = d_output_t0[layer];
	}
	else{
		if (useProbabilities)
			d_output = d_output_ptn[layer];
		else
			d_output = d_output_tn[layer];
	}
	//Matrix multiplication
	hipDeviceSynchronize();
	hipblasSgemm('n','n',batchSize,inputs,outputs,
			1.f,d_output,batchSize,
			d_weights[layer],outputs,
			0.f,d_input_p,batchSize);
	
	//probabilities kernel
	probabilities<<<numberOfBlocks,blockSize>>>(d_input_p, d_inputBiases[layer], inputBatchSize);
	
	//cutoff kernel
	cutoff<<<numberOfBlocks,blockSize>>>(d_input_p, d_input, d_randomNumbers, inputBatchSize);

};

void RBM::pushUp(int layer, bool input_t0, bool output_t0, bool useProbabilities){

	//Basic variables
	
	int inputs = layerSizes[layer]+labelSizes[layer];
	int outputs = layerSizes[layer+1];
	int outputBatchSize = outputs * batchSize;
	int numberOfBlocks = outputBatchSize/blockSize + (outputBatchSize%blockSize == 0?0:1);

	//device pointers
	float* d_input;
	float* d_output_p;
	float* d_output;
	if (input_t0){
		if (useProbabilities)
			d_input = d_input_pt0[layer];
		else
			d_input = d_input_t0[layer];	
	}
	else{
		if (useProbabilities)
			d_input = d_input_ptn[layer];
		else
			d_input = d_input_tn[layer];	
	}
	
	if (output_t0){
		d_output_p = d_output_pt0[layer];
		d_output = d_output_t0[layer];
	}
	else{
		d_output_p = d_output_ptn[layer];
		d_output = d_output_tn[layer];
	}
	//Matrix multiplication
	hipDeviceSynchronize();
	hipblasSgemm('n','T',batchSize,outputs,inputs,
			1.f,d_input,batchSize,
			d_weights[layer],outputs,
			0.f,d_output_p,batchSize);
	
	//probabilities kernel
	probabilities<<<numberOfBlocks,blockSize>>>(d_output_p, d_outputBiases[layer], outputBatchSize);
	
	//cutoff kernel
	cutoff<<<numberOfBlocks,blockSize>>>(d_output_p, d_output, d_randomNumbers, outputBatchSize);

};

void RBM::alternatingGibbsSampling(int layer, int iterations, bool probabilisticInput, bool probabilisticOutput, bool startAtTop){

	// Push up the initial pattern, then down to the inputs
	if (!startAtTop)
		pushUp(layer, true, true, probabilisticInput);
	pushDown(layer, false, true, probabilisticOutput);
	//Cycle doing this
	for( int i=0 ; i<iterations-1 ; i++ )
	{
		pushUp(layer, false, false, probabilisticInput);
		pushDown(layer, false, false, probabilisticOutput);
	}
	//Final push up.	
	pushUp(layer, false, false, probabilisticInput);
	
};

void RBM::updateBiasesInLayer(int layer){
	int inputs = layerSizes[layer]+labelSizes[layer];
	int outputs = layerSizes[layer+1];
	int inputBatchSize = inputs * batchSize;
	int outputBatchSize = outputs * batchSize;

	int nBlocksForInBiases = inputs/blockSize + (inputs%blockSize == 0?0:1);
	int nBlocksForOutBiases = outputs/blockSize + (outputs%blockSize == 0?0:1);

	// Update the input biases
	biasesIncrease<<<nBlocksForInBiases,blockSize>>>(d_input_pt0[layer], d_inputBiases[layer], biasLearningRates[layer], inputBatchSize/batchSize);
	biasesDecrease<<<nBlocksForInBiases,blockSize>>>(d_input_ptn[layer], d_inputBiases[layer], biasLearningRates[layer], inputBatchSize/batchSize, 0.0);

	// Update the output biases
	biasesIncrease<<<nBlocksForOutBiases,blockSize>>>(d_output_pt0[layer], d_outputBiases[layer], biasLearningRates[layer], outputBatchSize/batchSize);
	biasesDecrease<<<nBlocksForOutBiases,blockSize>>>(d_output_ptn[layer], d_outputBiases[layer], biasLearningRates[layer], outputBatchSize/batchSize, 0.0);

};

void RBM::updateWeightsInLayer(int layer){
	int inputs = layerSizes[layer]+labelSizes[layer];
	int outputs = layerSizes[layer+1];
	// Update the weights
	hipblasSgemm('T','n',outputs,inputs,batchSize,
			learningRates[layer],d_output_pt0[layer],batchSize,
			d_input_pt0[layer],batchSize,
			1.f,d_weights[layer],outputs);
	//checkError(cublasGetError());

	hipblasSgemm('T','n',outputs,inputs,batchSize,
			-learningRates[layer],d_output_ptn[layer],batchSize,
			d_input_ptn[layer],batchSize,
			weightDecay[layer],d_weights[layer],outputs);

};

void RBM::updateWeights(){

	int topRequiredLayer=0;

	// We need to know the top layer with a learning rate
	for( int layer=0 ; layer<numberOfNeuronLayers ; layer++ )
	{
		if( learningRates[layer]!=0.0 )
		{
			topRequiredLayer=layer;
		}
	}

	for( int layer=0 ; layer<topRequiredLayer; layer++ ){
		if( learningRates[layer]==0 ){
			pushUp(layer, true, true, true);
		}
		else{
			alternatingGibbsSampling(layer, CDSamples);
			updateWeightsInLayer(layer);
		}
	}

};

void RBM::setInputPattern(){
	//inputSource->nextInput(d_input_pt0[0], layerSizes[0]+labelSizes[0]);
};

void RBM::learningIteration(){
	
	setInputPattern();
	updateWeights();
	parameterUpdater->updateParameters(this);
};
