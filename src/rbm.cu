#include "hip/hip_runtime.h"
/*
 *   This file is part of RBM-on-GPU.
 *
 *   Foobar is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   Foobar is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with RBM-on-GPU.  If not, see <http://www.gnu.org/licenses/>.
 */

// Need to cut down on the includes, these are the max required, not sure
// which are needed
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include  <sys/timeb.h>
#include <allegro.h>

//mmap stuff
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/mman.h>


/* Includes, cuda */
#include "hipblas.h"
#include "kernels.cu"

