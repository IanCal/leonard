
#include <hip/hip_runtime.h>

/* Matrix size */
#define N  (1000)
#define blockSize (512)
#define WIDTH   800
#define HEIGHT  800
#define BATCHSIZE 512
#define WEIGHTDECAY 1.0

//need to add temperature to this really for sim allealing.
__global__ void probabilities( float* neurons, float* biases, int maxLength){
	
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<maxLength){
		neurons[idx] = 1./(1.+exp(-biases[idx/BATCHSIZE]-neurons[idx]));
		//neurons[idx] = 0.5+tanh(biases[idx/BATCHSIZE]+neurons[idx]);
	}
};

__global__ void softmax(float* in, float* out){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	in[idx]=expf(in[idx]);
	out[idx%BATCHSIZE]+=in[idx];
};
__global__ void arrayDivide(float* in, float* out){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (out[idx%BATCHSIZE]>0.)
		in[idx]/=out[idx%BATCHSIZE];
};
/*
__global__ void roulette(float* in){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
//	if (out[idx/numLabels]>0.)
//		in[idx]/=out[idx/numLabels];
};
*/
__global__ void biasesIncrease(float* in, float* out, float learningRate, int maxLength){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	//out[idx]=0.f;
	if (idx<maxLength)
	for( int i=0 ; i<BATCHSIZE ; i++ )
	{
		out[idx]+=in[idx*BATCHSIZE + i]*learningRate;
	}	
};

__global__ void biasesDecrease(float* in, float* out, float learningRate, int maxLength, float sparsity){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	//out[idx]=0.f;
	if (idx<maxLength)
	for( int i=0 ; i<BATCHSIZE ; i++ )
	{
		out[idx]-=in[idx*BATCHSIZE + i]*learningRate + sparsity;
	}	
};

__global__ void cutoff( float* neurons_in, float* neurons_out, float* random, int maxLength){
    
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<maxLength)
	neurons_out[idx]= (random[idx] < neurons_in[idx]) ? 1. : 0.;
	//neurons_out[idx]=neurons_in[idx];
};
