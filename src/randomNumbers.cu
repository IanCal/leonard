
#include <hip/hip_runtime.h>
/*
LICENSE: this code is subject to the license listed at
http://www.amolf.nl/~vanmeel/mdgpu/download.html
Among other restrictions, this code is released under the GNU Public License (GPL).

Authors:
A. Arnold (original)
Kipton Barros (modifications)

----

Generate pseudo-random numbers using a linear congruential generator. The generated
random numbers are identical to those produced by the lrand48() provided by the
C standard library.

Usage:

// From host, allocate the Rand48 structure, pass it to CUDA, and release it.
// The random sequence is persistent across CUDA kernel calls.

void hostFunction() {
    rng = new Rand48();
    rng->init(GRID_DIM*BLOCK_DIM, SEED);
    cudaFunction1 <<<GRID_DIM, BLOCK_DIM, sharedMem>>> (*rng);
    cudaFunction2 <<<GRID_DIM, BLOCK_DIM, sharedMem>>> (*rng);
    rng->destroy();
    delete rng;
}

// From CUDA, load the random state from device memory into local registers,
// generate random numbers, and finally store state back to device memory.
// Note that the random state, rng, is stored in registers, and is being updated
// with each device call.

__global__ void cudaFunction1(Rand48 rng) {
    rand48_loadState(rng);
    ...
    rand48_nextInt(rng);
    ...
    rand48_storeState(rng);
}

*/


struct Rand48 {
    // strided iteration constants (48-bit, distributed on 2x 24-bit)
    uint2 A, C;
    // CUDA array -- random numbers for all threads
    uint2 *state;
    // random number for a single thread (used by CUDA device functions only)
    uint2 state0;
    
    // magic constants for rand48
    static const unsigned long long a = 0x5DEECE66DLL, c = 0xB;
    
    void init(int nThreads, int seed) {
        uint2* seeds = new uint2[ nThreads ];
        
        hipMalloc((void**) &state, sizeof(uint2)*nThreads);
        
        // calculate strided iteration constants
        unsigned long long A, C;
        A = 1LL; C = 0LL;
        for (unsigned int i = 0; i < nThreads; ++i) {
            C += A*c;
            A *= a;
        }
        this->A.x = A & 0xFFFFFFLL;
        this->A.y = (A >> 24) & 0xFFFFFFLL;
        this->C.x = C & 0xFFFFFFLL;
        this->C.y = (C >> 24) & 0xFFFFFFLL;
        
        // prepare first nThreads random numbers from seed
        unsigned long long x = (((unsigned long long)seed) << 16) | 0x330E;
        for (unsigned int i = 0; i < nThreads; ++i) {
            x = a*x + c;
            seeds[i].x = x & 0xFFFFFFLL;
            seeds[i].y = (x >> 24) & 0xFFFFFFLL;
        }
        
        hipMemcpy(state, seeds, sizeof(uint2)*nThreads, hipMemcpyHostToDevice);
        
        delete[] seeds;
    }
    
    void destroy() {
        hipFree((void*) state);
    }
};

__device__ inline void rand48_loadState(Rand48 &r) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    r.state0 = r.state[i];
}

__device__ inline void rand48_storeState(Rand48 &r) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    r.state[i] = r.state0;
}

__device__ inline void rand48_iterate(Rand48 &r) {
    // state0 is 2x 24bit to handle overflows optimally, i.e.
    // in one operation.
    
    // the multiplication commands however give the low and hi 32 bit,
    // which have to be converted as follows:
    // 48bit in bytes = ABCD EF (space marks 32bit boundary)
    // R0             = ABC
    // R1             =    D EF
    
    unsigned int R0, R1;
    
    // low 24-bit multiplication
    const unsigned int lo00 = __umul24(r.state0.x, r.A.x);
    const unsigned int hi00 = __umulhi(r.state0.x, r.A.x);
    
    // 24bit distribution of 32bit multiplication results
    R0 = (lo00 & 0xFFFFFF);
    R1 = (lo00 >> 24) | (hi00 << 8);
    
    R0 += r.C.x; R1 += r.C.y;
    
    // transfer overflows
    R1 += (R0 >> 24);
    R0 &= 0xFFFFFF;
    
    // cross-terms, low/hi 24-bit multiplication
    R1 += __umul24(r.state0.y, r.A.x);
    R1 += __umul24(r.state0.x, r.A.y);
    
    R1 &= 0xFFFFFF;
    
    r.state0 = make_uint2(R0, R1);
}

__device__ inline int rand48_nextInt(Rand48 &r) {
    // get upper 31 (!) bits of the 2x 24bits
    int res = ( r.state0.x >> 17 ) | ( r.state0.y << 7 );
    rand48_iterate(r);
    return res;
}

// returns a float in the range [0, 1)
__device__ inline int rand48_nextFloat(Rand48 &r) {
    // use only upper 24 bits since floating point has 24 bit significand
    // (ref: Java random documentation)
    float res = r.state0.y / (float)(1<<24);
    rand48_iterate(r);
    return res;
}
